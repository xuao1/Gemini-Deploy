#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void testKernel(int *a) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    a[idx] = a[idx] + 1;
}

int main() {
    int *d_a;
    int size = 256 * sizeof(int);
    int *a = (int*)malloc(size);

    for (int i = 0; i < 256; i++) {
        a[i] = i;
    }

    hipMalloc((void **)&d_a, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    testKernel<<<1, 256>>>(d_a);

    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");

    hipFree(d_a);
    free(a);
    return 0;
}
