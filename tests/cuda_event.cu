#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipEvent_t startEvent, stopEvent;
    hipError_t error;

    // 创建 CUDA 事件
    error = hipEventCreate(&startEvent);
    if (error != hipSuccess) {
        std::cerr << "Failed to create start event: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    error = hipEventCreate(&stopEvent);
    if (error != hipSuccess) {
        std::cerr << "Failed to create stop event: " << hipGetErrorString(error) << std::endl;
        hipEventDestroy(startEvent); // 销毁已创建的事件
        return -1;
    }

    std::cout << "CUDA events created successfully." << std::endl;

    // 销毁事件
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return 0;
}
