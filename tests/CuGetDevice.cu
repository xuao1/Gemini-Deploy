#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    hipError_t res;
    hipDevice_t device;
    hipCtx_t context;

    // 初始化 CUDA
    res = hipInit(0);
    if (res != hipSuccess) {
        printf("hipInit failed: res = %d\n", res);
        return -1;
    }

    // 获取第一个可用的 CUDA 设备
    res = hipDeviceGet(&device, 0);
    if (res != hipSuccess) {
        printf("hipDeviceGet failed: res = %d\n", res);
        return -1;
    }

    // 创建一个上下文
    res = hipCtxCreate(&context, 0, device);
    if (res != hipSuccess) {
        printf("hipCtxCreate failed: res = %d\n", res);
        return -1;
    }

    // 获取当前的 CUDA 设备
    res = hipCtxGetDevice(&device);
    if (res != hipSuccess) {
        printf("hipCtxGetDevice failed: res = %d\n", res);
        return -1;
    }

    // 输出设备 ID
    printf("Device ID: %d\n", device);

    // 销毁上下文
    hipCtxDestroy(context);

    return 0;
}
