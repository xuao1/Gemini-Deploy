#include <stdio.h>
#include <hip/hip_runtime.h>


int main() {
    hipError_t res;
    hipDevice_t device;

    // 初始化 CUDA
    res = hipInit(0);
    if (res != hipSuccess) {
        printf("hipInit failed: res = %d\n", res);
        return -1;
    }

    // 获取当前的 CUDA 设备
    res = hipCtxGetDevice(&device);
    if (res != hipSuccess) {
        printf("hipCtxGetDevice failed: res = %d\n", res);
        return -1;
    }

    // 输出设备 ID
    printf("Device ID: %d\n", device);

    return 0;
}
